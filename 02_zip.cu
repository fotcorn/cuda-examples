#include <hip/hip_runtime.h>
#include <iostream>

__global__ void zip(const int *inA, const int *inB, int *out, const size_t n) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = inA[idx] + inB[idx];
    }
}

#define CUDA_CHECK(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(error) << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
} while (0)

int main() {
    // Check if there is a GPU available.
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

    // Initialize the input array on the host.
    const size_t n = 100;
    int hostInA[n];
    int hostInB[n];
    int hostOut[n];
    for (size_t i = 0; i < n; i++) {
        hostInA[i] = i;
        hostInB[i] = i + 10;
    }

    // Allocate memory on the device for both input and output arrays, which have the same size.
    int *deviceInA, *deviceInB, *deviceOut;
    CUDA_CHECK(hipMalloc((void **)&deviceInA, n * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&deviceInB, n * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&deviceOut, n * sizeof(int)));

    // Copy the input array from the host to the device.
    CUDA_CHECK(hipMemcpy(deviceInA, hostInA, n * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceInB, hostInB, n * sizeof(int), hipMemcpyHostToDevice));

    // Define the number of threads per block and the number of blocks.
    const int threadsPerBlock = 256; // Standard value, good balance between resource usage (registers and shared memory) and performance.
    const int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    std::cout << "blocks: " << blocks << ", threadsPerBlock: " << threadsPerBlock << std::endl;
    zip<<<blocks, threadsPerBlock>>>(deviceInA, deviceInB, deviceOut, n);
    CUDA_CHECK(hipGetLastError()); // Check for any errors in the kernel launch.
    CUDA_CHECK(hipDeviceSynchronize()); // Wait for all threads to complete.

    // Copy the output array from the device to the host.
    CUDA_CHECK(hipMemcpy(hostOut, deviceOut, n * sizeof(int), hipMemcpyDeviceToHost));

    // Print the output array.
    for (size_t i = 0; i < n; i++) {
        std::cout << hostOut[i] << " ";
    }
    std::cout << std::endl;

    // Release the allocated memory on the device.
    CUDA_CHECK(hipFree(deviceInA));
    CUDA_CHECK(hipFree(deviceInB));
    CUDA_CHECK(hipFree(deviceOut));

    return 0;
}
