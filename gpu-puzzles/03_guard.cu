#include <hip/hip_runtime.h>
#include <iostream>

__global__ void map(const int *in, int *out, const size_t n) {
    const size_t idx = threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx] + 10;
    }
}

#define CUDA_CHECK(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(error) << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
} while (0)

int main() {
    // Check if there is a GPU available.
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

    // Initialize the input array on the host.
    const size_t n = 100;
    int hostIn[n];
    int hostOut[n];
    for (size_t i = 0; i < n; i++) {
        hostIn[i] = i;
    }

    // Allocate memory on the device for both input and output arrays, which have the same size.
    int *deviceIn, *deviceOut;
    CUDA_CHECK(hipMalloc((void **)&deviceIn, n * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&deviceOut, n * sizeof(int)));

    // Copy the input array from the host to the device.
    CUDA_CHECK(hipMemcpy(deviceIn, hostIn, n * sizeof(int), hipMemcpyHostToDevice));

    // Define the number of threads per block and the number of blocks.
    // 256 Standard value, good balance between resource usage (registers and shared memory) and performance.
    // Also, it's bigger than n, so we only need one block.
    const int threadsPerBlock = 256; 

    std::cout << "blocks: " << 1 << ", threadsPerBlock: " << threadsPerBlock << std::endl;
    map<<<1, threadsPerBlock>>>(deviceIn, deviceOut, n);
    CUDA_CHECK(hipGetLastError()); // Check for any errors in the kernel launch.
    CUDA_CHECK(hipDeviceSynchronize()); // Wait for all threads to complete.

    // Copy the output array from the device to the host.
    CUDA_CHECK(hipMemcpy(hostOut, deviceOut, n * sizeof(int), hipMemcpyDeviceToHost));

    // Print the output array.
    for (size_t i = 0; i < n; i++) {
        std::cout << hostOut[i] << " ";
    }
    std::cout << std::endl;

    // Release the allocated memory on the device.
    CUDA_CHECK(hipFree(deviceIn));
    CUDA_CHECK(hipFree(deviceOut));

    return 0;
}
