#include <hip/hip_runtime.h>
#include <iostream>

__global__ void convolution1d(const int *a, const int *b, int *out, const size_t n, const size_t convSize) {
    extern __shared__ int shared[];

    // Global index into the full array.
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    // Local index into the shared array. Shared memory is shared across threads in the same block.
    const size_t localIdx = threadIdx.x;

    if (localIdx < n) {
        shared[localIdx] = a[idx];
    }
    if (localIdx < convSize) {
        shared[n + localIdx] = b[idx];
    }

    __syncthreads();

    int sum = 0;
    for (size_t i = 0; i < convSize; i++) {
        if (localIdx + i < n) {
            sum += shared[localIdx + i] * shared[i];
        }
    }
    out[idx] = sum;
}


#define CUDA_CHECK(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(error) << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
} while (0)


int main() {
    // Check if there is a GPU available.
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

    // Initialize the input array on the host.
    const size_t n = 6;
    const size_t convSize = 3;
    int hostInA[n];
    int hostInB[convSize];
    int hostOut[n];
    for (size_t i = 0; i < n; i++) {
        hostInA[i] = i;
        std::cout << hostInA[i] << " ";
    }
    std::cout << std::endl;
    for (size_t i = 0; i < convSize; i++) {
        hostInB[i] = i;
        std::cout << hostInB[i] << " ";
    }
    std::cout << std::endl;

    // Allocate memory on the device for both input and output arrays, which have the same size.
    int *deviceInA, *deviceInB, *deviceOut;
    CUDA_CHECK(hipMalloc((void **)&deviceInA, n * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&deviceInB, convSize * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&deviceOut, n *sizeof(int)));

    // Copy the input array from the host to the device.
    CUDA_CHECK(hipMemcpy(deviceInA, hostInA, n * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(deviceInB, hostInB, convSize * sizeof(int), hipMemcpyHostToDevice));

    // Define the number of threads per block and the number of blocks.
    const int threadsPerBlock = 256; // Standard value, good balance between resource usage (registers and shared memory) and performance.
    const int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    std::cout << "blocks: " << blocks << ", threadsPerBlock: " << threadsPerBlock << std::endl;
    convolution1d<<<blocks, threadsPerBlock, (n + convSize) * sizeof(int)>>>(deviceInA, deviceInB, deviceOut, n, convSize);
    CUDA_CHECK(hipGetLastError()); // Check for any errors in the kernel launch.
    CUDA_CHECK(hipDeviceSynchronize()); // Wait for all threads to complete.

    // Copy the output array from the device to the host.
    CUDA_CHECK(hipMemcpy(hostOut, deviceOut, n * sizeof(int), hipMemcpyDeviceToHost));

    // Print the output array.
    for (size_t i = 0; i < n; i++) {
        std::cout << hostOut[i] << " ";
    }
    std::cout << std::endl;

    // Release the allocated memory on the device.
    CUDA_CHECK(hipFree(deviceInA));
    CUDA_CHECK(hipFree(deviceInB));
    CUDA_CHECK(hipFree(deviceOut));

    return 0;
}

