#include <hip/hip_runtime.h>
#include <iostream>

__global__ void map(const int *in, int *out, const size_t n) {
    const size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = in[idx] + 10;
    }
}

#define CUDA_CHECK(call) do { \
    hipError_t error = call; \
    if (error != hipSuccess) { \
        std::cerr << "CUDA error " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(error) << std::endl; \
        std::exit(EXIT_FAILURE); \
    } \
} while (0)

int main() {
    // Check if there is a GPU available.
    hipDeviceProp_t prop;
    CUDA_CHECK(hipGetDeviceProperties(&prop, 0));

    // Initialize the input array on the host.
    const size_t n = 1353;
    int hostIn[n];
    int hostOut[n];
    for (size_t i = 0; i < n; i++) {
        hostIn[i] = i;
    }

    // Allocate memory on the device for both input and output arrays, which have the same size.
    int *deviceIn, *deviceOut;
    CUDA_CHECK(hipMalloc((void **)&deviceIn, n * sizeof(int)));
    CUDA_CHECK(hipMalloc((void **)&deviceOut, n * sizeof(int)));

    // Copy the input array from the host to the device.
    CUDA_CHECK(hipMemcpy(deviceIn, hostIn, n * sizeof(int), hipMemcpyHostToDevice));

    // Define the number of threads per block and the number of blocks.
    const int threadsPerBlock = 256;
    const int blocks = (n + threadsPerBlock - 1) / threadsPerBlock;

    std::cout << "blocks: " << blocks << ", threadsPerBlock: " << threadsPerBlock << std::endl;
    map<<<blocks, threadsPerBlock>>>(deviceIn, deviceOut, n);
    CUDA_CHECK(hipGetLastError()); // Check for any errors in the kernel launch.
    CUDA_CHECK(hipDeviceSynchronize()); // Wait for all threads to complete.

    // Copy the output array from the device to the host.
    CUDA_CHECK(hipMemcpy(hostOut, deviceOut, n * sizeof(int), hipMemcpyDeviceToHost));

    // Print the output array.
    for (size_t i = 0; i < n; i++) {
        std::cout << hostOut[i] << " ";
    }
    std::cout << std::endl;

    // Release the allocated memory on the device.
    CUDA_CHECK(hipFree(deviceIn));
    CUDA_CHECK(hipFree(deviceOut));

    return 0;
}
